#include "hip/hip_runtime.h"
/*
 * Test.cpp
 *
 *  Created on: Dec 23, 2016
 *      Author: zys
 */
#include "Resource.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <pthread.h>
using namespace std;
#include <omp.h>

#include "ZeroTest.cuh"

__global__ void myKernel(int *d_o)
{
    long delay=100000000;
    while(delay>0){
    	delay--;
    }

    for(int i=0;i<1024*1024;++i){
    	d_o[i]=i;
    }
}

void testDgemmWithVec(){

	ostringstream oss;
	int fragmentNumber=3;
	double alpha = 1.0;
	double beta = 0.0;
	hipblasHandle_t handles[fragmentNumber];
	hipStream_t streams[fragmentNumber];
	for(int i=0;i<fragmentNumber;++i){
		checkCublasErrors(hipblasCreate(&handles[i]));
		hipStreamCreate(&streams[i]);
	}


	int M = 150;  //batch_size
	int N = 256*36; //in
	int Q = 4096; //out

	long size_A = (M * N); //input_data 3*2
	long size_B = (N * Q); //weight 2*1
	long size_C = (M * Q); //output_data 3*1

//	double *A,*B,*C;
//	A=(double*)malloc(size_A*sizeof(double));
//	oss.str("");
//	for(int i=0;i<size_A;++i){
//		A[i]=rand()%3;
//		oss<<A[i]<<" ";
//	}
//	LOG(INFO)<<"A:"<<oss.str();
//	sleep(1);
//	B=(double*)malloc(size_B*sizeof(double));
//	oss.str("");
//	for(int i=0;i<size_B;i++){
//		B[i]=rand()%2;
//		oss<<B[i]<<" ";
//	}
//	LOG(INFO)<<"B:"<<oss.str();
//
//	C=(double*)malloc(size_C*sizeof(double));
//	double *d_A,*d_B,*d_C;
//	checkCudaErrors(hipMalloc(&d_A, size_A * sizeof(double)));
//	checkCudaErrors(hipMalloc(&d_B, size_B * sizeof(double)));
//	checkCudaErrors(hipMemcpy((void*)d_A,(void*)A,size_A*sizeof(double),hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy((void*)d_B,(void*)B,size_B*sizeof(double),hipMemcpyHostToDevice));
//	checkCudaErrors(hipMalloc(&d_C, size_C * sizeof(double)));
//	checkCublasErrors(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha, d_B, Q, d_A, N, &beta, d_C, Q));
//
//	checkCudaErrors(hipMemcpy((void*)C,(void*)d_C,size_C*sizeof(double),hipMemcpyDeviceToHost));
//	oss.str("");
//	for(int i=0;i<size_C;++i){
//		oss<<C[i]<<" ";
//	}
//	LOG(INFO)<<oss.str();



	vector<double*> d_InDataVec;
	vector<double*> d_OutDataVec;

	double* d_Weight=NULL;
	checkCudaErrors(hipMalloc((void**)&d_Weight,size_B* sizeof(double)));
	double* weight=NULL;
	weight=(double*)malloc(size_B*sizeof(double));
//	srand(time(NULL));
//	oss.str("");
//	for(int i=0;i<size_B;++i){
//		weight[i]=rand()%3;
//		oss<<weight[i]<<" ";
//	}
//	LOG(INFO)<<"Weight:"<<oss.str();
	checkCudaErrors(hipMemcpy((void*)d_Weight,(void*)weight,size_B*sizeof(double),hipMemcpyHostToDevice));

	double* in=NULL;
	in=(double*)malloc(size_A/fragmentNumber*sizeof(double));
	for(int i=0;i<fragmentNumber;++i){
		double* d_in=NULL;
		double* d_out=NULL;
//		sleep(1);
//		srand(time(NULL));
//		oss.str("");
//		for(int j=0;j<size_A/fragmentNumber;++j){
//			in[j]=rand()%3;
//			oss<<in[j]<<" ";
//		}
//		LOG(INFO)<<"InData:"<<oss.str();

		checkCudaErrors(hipMalloc((void**)&d_in, size_A/fragmentNumber * sizeof(double)));
		checkCudaErrors(hipMemcpy((void*)d_in,(void*)in,size_A/fragmentNumber*sizeof(double),hipMemcpyHostToDevice));
		checkCudaErrors(hipMalloc((void**)&d_out, size_C/fragmentNumber * sizeof(double)));
		d_InDataVec.push_back(d_in);
		d_OutDataVec.push_back(d_out);
	}

	for(int i=0;i<fragmentNumber;++i){
		hipblasSetStream(handles[i],streams[i]);
		checkCublasErrors(hipblasDgemm(handles[i], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M/fragmentNumber, N, &alpha, d_Weight, Q, d_InDataVec[i], N, &beta, d_OutDataVec[i], Q));

	}

//	double* out=NULL;
//	out=(double*)malloc(size_C/fragmentNumber*sizeof(double));
//	checkCudaErrors(hipMemcpy((void*)out,(void*)d_OutDataVec[0],size_C/fragmentNumber*sizeof(double),hipMemcpyDeviceToHost));
//	oss.str("");
//	for(int i=0;i<size_C/fragmentNumber;++i){
//		oss<<out[i]<<" ";
//	}
//	LOG(INFO)<<"outData:"<<oss.str();
//
//	checkCudaErrors(hipMemcpy((void*)out,(void*)d_OutDataVec[1],size_C/fragmentNumber*sizeof(double),hipMemcpyDeviceToHost));
//	oss.str("");
//	for(int i=0;i<size_C/fragmentNumber;++i){
//		oss<<out[i]<<" ";
//	}
//	LOG(INFO)<<"outData:"<<oss.str();
//	free(out);
//

	free(in);
	free(weight);

	hipFree(d_Weight);
	for(int i=0;i<fragmentNumber;++i){
		hipFree(d_InDataVec[i]);
		hipFree(d_OutDataVec[i]);
		hipStreamDestroy(streams[i]);
		checkCublasErrors(hipblasDestroy(handles[i]));

	}

	hipDeviceReset();

//	free(A);
//	free(B);
//	free(C);
//	hipFree(d_B);
//	hipFree(d_A);
//	hipFree(d_C);
}

void testDgemm(){

	ostringstream oss;
	int fragmentNumber=3;
	double alpha = 1.0;
	double beta = 0.0;
	hipblasHandle_t handles[fragmentNumber];
	hipStream_t streams[fragmentNumber];
	for(int i=0;i<fragmentNumber;++i){
		checkCublasErrors(hipblasCreate(&handles[i]));
		hipStreamCreate(&streams[i]);
	}


	int M = 150;  //batch_size
	int N = 256*36; //in
	int Q = 4096; //out

	long size_A = (M * N); //input_data 3*2
	long size_B = (N * Q); //weight 2*1
	long size_C = (M * Q); //output_data 3*1

//	double *A,*B,*C;
//	A=(double*)malloc(size_A*sizeof(double));
//	oss.str("");
//	for(int i=0;i<size_A;++i){
//		A[i]=rand()%3;
//		oss<<A[i]<<" ";
//	}
//	LOG(INFO)<<"A:"<<oss.str();
//	sleep(1);
//	B=(double*)malloc(size_B*sizeof(double));
//	oss.str("");
//	for(int i=0;i<size_B;i++){
//		B[i]=rand()%2;
//		oss<<B[i]<<" ";
//	}
//	LOG(INFO)<<"B:"<<oss.str();
//
//	C=(double*)malloc(size_C*sizeof(double));
//	double *d_A,*d_B,*d_C;
//	checkCudaErrors(hipMalloc(&d_A, size_A * sizeof(double)));
//	checkCudaErrors(hipMalloc(&d_B, size_B * sizeof(double)));
//	checkCudaErrors(hipMemcpy((void*)d_A,(void*)A,size_A*sizeof(double),hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy((void*)d_B,(void*)B,size_B*sizeof(double),hipMemcpyHostToDevice));
//	checkCudaErrors(hipMalloc(&d_C, size_C * sizeof(double)));
//	checkCublasErrors(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha, d_B, Q, d_A, N, &beta, d_C, Q));
//
//	checkCudaErrors(hipMemcpy((void*)C,(void*)d_C,size_C*sizeof(double),hipMemcpyDeviceToHost));
//	oss.str("");
//	for(int i=0;i<size_C;++i){
//		oss<<C[i]<<" ";
//	}
//	LOG(INFO)<<oss.str();



	double* d_InData;
	double* d_OutData;
	double* inData=NULL;
	inData=(double*)malloc(size_A*sizeof(double));
	srand(time(NULL));
	oss.str("");
	for(int i=0;i<size_A;++i){
		inData[i]=rand()%3;
//		oss<<inData[i]<<" ";
//
//		if(((i+1)%N)==0){
//			oss<<"\n";
//		}
	}
	//LOG(INFO)<<"InData:"<<oss.str();
	checkCudaErrors(hipMalloc((void**)&d_InData, size_A* sizeof(double)));
	checkCudaErrors(hipMemcpy((void*)d_InData,(void*)inData,size_A*sizeof(double),hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&d_OutData, size_C * sizeof(double)));
	double* d_Weight=NULL;
	checkCudaErrors(hipMalloc((void**)&d_Weight,size_B* sizeof(double)));
	double* weight=NULL;
	weight=(double*)malloc(size_B*sizeof(double));
	srand(time(NULL));
	oss.str("");
	for(int i=0;i<size_B;++i){
		weight[i]=rand()%3;
		oss<<weight[i]<<" ";
	}
	//LOG(INFO)<<"Weight:"<<oss.str();
	checkCudaErrors(hipMemcpy((void*)d_Weight,(void*)weight,size_B*sizeof(double),hipMemcpyHostToDevice));

	double* out=NULL;
	out=(double*)malloc(size_C/fragmentNumber*sizeof(double));
	for(int i=0;i<fragmentNumber;++i){
		hipblasSetStream(handles[i],streams[i]);
	}
	for(int i=0;i<fragmentNumber;++i){
		checkCublasErrors(hipblasDgemm(handles[i], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M/fragmentNumber, N, &alpha, d_Weight, Q, (d_InData+i*size_A/fragmentNumber), N, &beta, (d_OutData+i*size_C/fragmentNumber), Q));
//		checkCudaErrors(hipMemcpy((void*)out,(void*)(d_OutData+i*size_C/fragmentNumber),size_C/fragmentNumber*sizeof(double),hipMemcpyDeviceToHost));
//		oss.str("");
//		for(int j=0;j<size_C/fragmentNumber;++j){
//			oss<<out[j]<<" ";
//		}
//		LOG(INFO)<<"outData:"<<oss.str();
	}

//
//	checkCudaErrors(hipMemcpy((void*)out,(void*)(d_OutData+0*size_C/fragmentNumber),size_C/fragmentNumber*sizeof(double),hipMemcpyDeviceToHost));
//	oss.str("");
//	for(int i=0;i<size_C/fragmentNumber;++i){
//		oss<<out[i]<<" ";
//	}
//	LOG(INFO)<<"outData:"<<oss.str();
//
//	checkCudaErrors(hipMemcpy((void*)out,(void*)(d_OutData+1*size_C/fragmentNumber),size_C/fragmentNumber*sizeof(double),hipMemcpyDeviceToHost));
//	oss.str("");
//	for(int i=0;i<size_C/fragmentNumber;++i){
//		oss<<out[i]<<" ";
//	}
//	LOG(INFO)<<"outData:"<<oss.str();
	free(out);
	free(weight);
	hipFree(d_InData);
	hipFree(d_OutData);
	hipFree(d_Weight);
	for(int i=0;i<fragmentNumber;++i){
		hipStreamDestroy(streams[i]);
		checkCublasErrors(hipblasDestroy(handles[i]));
	}
	hipDeviceReset();

//	free(A);
//	free(B);
//	free(C);
//	hipFree(d_B);
//	hipFree(d_A);
//	hipFree(d_C);
}
void testKernelWithStreams() {

	int nkernels = 32;
	int nStreams = nkernels;
    int nbytes=nkernels*1024*1024*sizeof(int)*10;
    int *d_a = 0;
    checkCudaErrors(hipMalloc((void **)&d_a, nbytes));

	hipStream_t streams[nStreams];
	for (int i = 0; i < nStreams; ++i) {
		hipStreamCreate(&streams[i]);
	}

	double startTime = omp_get_wtime();
	for (int i=0; i<nkernels; ++i){
		dim3 gridSize(64,1);
		dim3 blockSize(2,1);
		myKernel<<<gridSize,blockSize,0,streams[i]>>>(&d_a[i]);
	}
	hipDeviceSynchronize();
	double endTime = omp_get_wtime();
	cout << "Runtime:" << endTime - startTime << endl;
    hipFree(d_a);
	for (int i = 0; i < nStreams; ++i) {
		hipStreamDestroy(streams[i]);
	}
}

void testGemmsOnSameDevice(){
    int nkernels = 2;               // number of concurrent kernels
    int nstreams = nkernels;    // use one more stream than concurrent kernel
    int cuda_device = 0;

    float elapsed_time;   // timing variables
 //   printf("[%s] - Starting...\n", argv[0]);

    //fc6
//    int M = 40;  //batch_size
//    int N = 256*6*6; //in
//    int Q = 4096; //out

    //fc7
    int M = 40;  //batch_size
	int N = 4096; //in
	int Q = 4096; //out


//  int M = 128;  //batch_size
//	int N = 128; //in
//	int Q = 128; //out


	long size_A = (M * N); //input_data
	long size_B = (N * Q); //weight
	long size_C = (M * Q); //output_data

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDevice(&cuda_device));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

    if ((deviceProp.concurrentKernels == 0))
    {
        printf("> GPU does not support concurrent kernel execution\n");
        printf("  CUDA kernel runs will be serialized\n");
    }

    printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));
	hipblasHandle_t *cublasHandle=(hipblasHandle_t *)malloc(nstreams * sizeof(hipblasHandle_t));

    for (int i = 0; i < nstreams; i++)
    {
        checkCudaErrors(hipStreamCreate(&(streams[i])));
        checkCublasErrors(hipblasCreate(&cublasHandle[i]));
    }

	double *d_A,*d_B,*d_C;
	checkCudaErrors(hipMalloc(&d_A, size_A * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_B, size_B * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_C, size_C * sizeof(double)));
	double *d_A1,*d_B1,*d_C1;
	checkCudaErrors(hipMalloc(&d_A1, size_A * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_B1, size_B * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_C1, size_C * sizeof(double)));

	double alpha = 1.0;
	double beta = 0.0;

	double alpha1 = 1.0;
	double beta1 = 0.0;
    // create CUDA event handles
    hipEvent_t start_event, stop_event;
    checkCudaErrors(hipEventCreate(&start_event));
    checkCudaErrors(hipEventCreate(&stop_event));
    checkCudaErrors(hipEventRecord(start_event, 0));

    checkCublasErrors(hipblasSetStream(cublasHandle[0],streams[0]));
    checkCublasErrors(hipblasDgemm(cublasHandle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha, d_B, Q, d_A, N, &beta, d_C, Q));
    checkCublasErrors(hipblasDgemm(cublasHandle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha, d_B, Q, d_A, N, &beta, d_C, Q));
    checkCublasErrors(hipblasDgemm(cublasHandle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha, d_B, Q, d_A, N, &beta, d_C, Q));
    checkCublasErrors(hipblasDgemm(cublasHandle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha, d_B, Q, d_A, N, &beta, d_C, Q));


    checkCublasErrors(hipblasSetStream(cublasHandle[1],streams[1]));
    checkCublasErrors(hipblasDgemm(cublasHandle[1], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha1, d_B1, Q, d_A1, N, &beta1, d_C1, Q));
    checkCublasErrors(hipblasDgemm(cublasHandle[1], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha1, d_B1, Q, d_A1, N, &beta1, d_C1, Q));
    checkCublasErrors(hipblasDgemm(cublasHandle[1], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha1, d_B1, Q, d_A1, N, &beta1, d_C1, Q));
    checkCublasErrors(hipblasDgemm(cublasHandle[1], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha1, d_B1, Q, d_A1, N, &beta1, d_C1, Q));

    checkCudaErrors(hipEventRecord(stop_event, 0));
    checkCudaErrors(hipEventSynchronize(stop_event));
    checkCudaErrors(hipEventElapsedTime(&elapsed_time, start_event, stop_event));
    cout<<"Time:"<<elapsed_time<<endl;

    hipDeviceSynchronize();

    // release resources
    for (int i = 0; i < nkernels; i++)
    {
        hipStreamDestroy(streams[i]);
        checkCublasErrors(hipblasDestroy(cublasHandle[i]));
    }

    free(streams);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_A1);
	hipFree(d_B1);
	hipFree(d_C1);

//    hipFree(d_X);
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

    printf("Test passed\n");
}

void testGemmsOnDiffDevice(){
    int nkernels = 2;               // number of concurrent kernels
    int nstreams = nkernels;    // use one more stream than concurrent kernel
    int cuda_device = 0;
    int nbytes=nkernels*1024*1024*sizeof(int)*10;


    //fc6
    int M = 100;  //batch_size
    int N = 256*6*6; //in
    int Q = 4096; //out

    //fc7
//    int M = 100;  //batch_size
//	int N = 4096; //in
//	int Q = 4096; //out

	long size_A = (M * N); //input_data
	long size_B = (N * Q); //weight
	long size_C = (M * Q); //output_data

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDevice(&cuda_device));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

    if ((deviceProp.concurrentKernels == 0))
    {
        printf("> GPU does not support concurrent kernel execution\n");
        printf("  CUDA kernel runs will be serialized\n");
    }

    printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // allocate and initialize an array of stream handles
//  hipStream_t *streams = (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));
	hipblasHandle_t *cublasHandle=(hipblasHandle_t *)malloc(nstreams * sizeof(hipblasHandle_t));

//    for (int i = 0; i < nstreams; i++)
//    {
//        checkCudaErrors(hipStreamCreate(&(streams[i])));
//        checkCublasErrors(hipblasCreate(&cublasHandle[i]));
//    }

	double *d_A,*d_B,*d_C;
	hipSetDevice(0);
	checkCudaErrors(hipMalloc(&d_A, size_A * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_B, size_B * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_C, size_C * sizeof(double)));
    checkCublasErrors(hipblasCreate(&cublasHandle[0]));
	double *d_A1,*d_B1,*d_C1;
	hipSetDevice(1);
	checkCudaErrors(hipMalloc(&d_A1, size_A * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_B1, size_B * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_C1, size_C * sizeof(double)));
    checkCublasErrors(hipblasCreate(&cublasHandle[1]));

    // allocate device memory
    int *d_a = 0;             // pointers to data and init value in the device memory
    checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
    int *a=(int*)malloc(nbytes);


	double alpha = 1.0;
	double beta = 0.0;

	double alpha1 = 1.0;
	double beta1 = 0.0;
    // create CUDA event handles
//    hipEvent_t start_event, stop_event;
//    checkCudaErrors(hipEventCreate(&start_event));
//    checkCudaErrors(hipEventCreate(&stop_event));
//    checkCudaErrors(hipEventRecord(start_event, 0));
	double startTime = omp_get_wtime();
	hipSetDevice(0);
    checkCublasErrors(hipblasDgemm(cublasHandle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha, d_B, Q, d_A, N, &beta, d_C, Q));
    checkCublasErrors(hipblasDgemm(cublasHandle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha, d_B, Q, d_A, N, &beta, d_C, Q));
    checkCublasErrors(hipblasDgemm(cublasHandle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha, d_B, Q, d_A, N, &beta, d_C, Q));
    checkCublasErrors(hipblasDgemm(cublasHandle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha, d_B, Q, d_A, N, &beta, d_C, Q));


	hipSetDevice(1);
    checkCublasErrors(hipblasDgemm(cublasHandle[1], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha1, d_B1, Q, d_A1, N, &beta1, d_C1, Q));
    checkCublasErrors(hipblasDgemm(cublasHandle[1], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha1, d_B1, Q, d_A1, N, &beta1, d_C1, Q));
    checkCublasErrors(hipblasDgemm(cublasHandle[1], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha1, d_B1, Q, d_A1, N, &beta1, d_C1, Q));
    checkCublasErrors(hipblasDgemm(cublasHandle[1], HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha1, d_B1, Q, d_A1, N, &beta1, d_C1, Q));


    hipDeviceSynchronize();
	double endTime = omp_get_wtime();
	cout << "Runtime:" << endTime - startTime << endl;
//	for (int i=0; i<nkernels; ++i)
//	{
//		dim3 gridSize(8,8);
//		dim3 blockSize(16,16);
//		myKernel<<<gridSize,blockSize,0,streams[i]>>>(&d_a[i]);
//	}
//    checkCudaErrors(hipEventRecord(stop_event, 0));
//    checkCudaErrors(hipEventSynchronize(stop_event));
//    checkCudaErrors(hipEventElapsedTime(&elapsed_time, start_event, stop_event));
//    cout<<"Time:"<<elapsed_time<<endl;
    // release resources
    for (int i = 0; i < nkernels; i++)
    {
        checkCublasErrors(hipblasDestroy(cublasHandle[i]));
    }

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_A1);
	hipFree(d_B1);
	hipFree(d_C1);
    hipFree(d_a);
    free(a);
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

    printf("Test passed\n");
}

void SetTensorDesc(hipdnnTensorDescriptor_t& tensorDesc, hipdnnDataType_t dataType, int n, int c, int h, int w) {
	const int nDims = 4;
	int dimA[nDims] = { n, c, h, w };
	int strideA[nDims] = { c * h * w, h * w, w, 1 };
	checkCUDNN(hipdnnSetTensorNdDescriptor(tensorDesc, dataType, 4, dimA, strideA));
}

void testConnvOnSameDevice(){

	int m_convAlgorithm;
	hipdnnDataType_t m_dataType;
	hipdnnHandle_t m_cudnnHandle[2];
	hipdnnTensorDescriptor_t m_srcTensorDesc[2], m_dstTensorDesc[2], m_biasTensorDesc[2];
	hipdnnFilterDescriptor_t m_filterDesc[2];
	hipdnnConvolutionDescriptor_t m_convDesc[2];

	m_dataType = HIPDNN_DATA_DOUBLE;
	m_convAlgorithm = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;

	checkCUDNN(hipdnnCreate(&m_cudnnHandle[0]));
	checkCUDNN(hipdnnCreateTensorDescriptor(&m_srcTensorDesc[0]));
	checkCUDNN(hipdnnCreateTensorDescriptor(&m_dstTensorDesc[0]));
	checkCUDNN(hipdnnCreateTensorDescriptor(&m_biasTensorDesc[0]));
	checkCUDNN(hipdnnCreateFilterDescriptor(&m_filterDesc[0]));
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&m_convDesc[0]));

	checkCUDNN(hipdnnCreate(&m_cudnnHandle[1]));
	checkCUDNN(hipdnnCreateTensorDescriptor(&m_srcTensorDesc[1]));
	checkCUDNN(hipdnnCreateTensorDescriptor(&m_dstTensorDesc[1]));
	checkCUDNN(hipdnnCreateTensorDescriptor(&m_biasTensorDesc[1]));
	checkCUDNN(hipdnnCreateFilterDescriptor(&m_filterDesc[1]));
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&m_convDesc[1]));

	int batchSize = 10;
	//conv2
	int kWidth = 5, kHeight = 5;
	int inMaps = 96, outMaps = 256;
	int inMapSize = 27, outMapSize = 27;

	//conv3
//	int kWidth = 3, kHeight = 3;
//	int inMaps = 256, outMaps = 384;
//	int inMapSize = 13, outMapSize = 13;

	int n, c, h, w;
	double* deviceInData;
	double* deviceOutData;
	double* deviceWeight;

	double* deviceInData1;
	double* deviceOutData1;
	double* deviceWeight1;

	n = batchSize;
	c = inMaps;
	h = inMapSize;
	w = inMapSize;
	SetTensorDesc(m_srcTensorDesc[0], m_dataType, n, c, h, w);
	SetTensorDesc(m_srcTensorDesc[1], m_dataType, n, c, h, w);
	hipMalloc(&deviceInData, n * c * h * w * sizeof(double));
	hipMalloc(&deviceInData1, n * c * h * w * sizeof(double));

	c = outMaps;
	h = outMapSize;
	w = outMapSize;
	SetTensorDesc(m_dstTensorDesc[0], m_dataType, n, c, h, w);
	SetTensorDesc(m_dstTensorDesc[1], m_dataType, n, c, h, w);
	hipMalloc(&deviceOutData, n * c * h * w * sizeof(double));
	hipMalloc(&deviceOutData1, n * c * h * w * sizeof(double));

	const int tensorDims = 4;
	const int filterDimA[tensorDims] = { outMaps, inMaps, kWidth, kHeight };
	checkCUDNN(hipdnnSetFilterNdDescriptor(m_filterDesc[0], m_dataType, HIPDNN_TENSOR_NCHW, tensorDims, filterDimA));
	checkCUDNN(hipdnnSetFilterNdDescriptor(m_filterDesc[1], m_dataType, HIPDNN_TENSOR_NCHW, tensorDims, filterDimA));

	const int convDims = 2;
	int padA[convDims] = { 0, 0 };
	int filterStrideA[convDims] = { 1, 1 };
	int upscaleA[convDims] = { 1, 1 };
	checkCUDNN(hipdnnSetConvolutionNdDescriptor(m_convDesc[0], convDims, padA, filterStrideA, upscaleA, HIPDNN_CROSS_CORRELATION, m_dataType));
	checkCUDNN(hipdnnSetConvolutionNdDescriptor(m_convDesc[1], convDims, padA, filterStrideA, upscaleA, HIPDNN_CROSS_CORRELATION, m_dataType));

	long weightSize = kWidth * kHeight * outMaps * inMaps;
	hipMalloc(&deviceWeight, weightSize * sizeof(double));
	hipMalloc(&deviceWeight1, weightSize * sizeof(double));

	size_t m_fwdDataSizeInBytes;
	void * m_fwdDataWorkSpace;
	void * m_fwdDataWorkSpace1;
	hipdnnConvolutionFwdAlgo_t algo = (hipdnnConvolutionFwdAlgo_t) m_convAlgorithm;

	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(m_cudnnHandle[0], m_srcTensorDesc[0], m_filterDesc[0], m_convDesc[0], m_dstTensorDesc[0], algo, &m_fwdDataSizeInBytes));
	if (m_fwdDataSizeInBytes != 0) {
		checkCudaErrors(hipMalloc(&m_fwdDataWorkSpace, m_fwdDataSizeInBytes));
		checkCudaErrors(hipMalloc(&m_fwdDataWorkSpace1, m_fwdDataSizeInBytes));
	}
	hipStream_t streams[2];
	for (int i = 0; i < 2; i++)
	{
		checkCudaErrors(hipStreamCreate(&(streams[i])));
	}


	double alpha = 1.0;
	double beta = 0.0;
	hipdnnSetStream(m_cudnnHandle[0],streams[0]);

	for(int i=0;i<4;i++){
		checkCUDNN(hipdnnConvolutionForward(m_cudnnHandle[0], //
					&alpha, //
					m_srcTensorDesc[0], //
					deviceInData, //the input data of current layer ,also is the output data of preLayer.
					m_filterDesc[0], //
					deviceWeight, //
					m_convDesc[0], //
					algo, //
					m_fwdDataWorkSpace, //
					m_fwdDataSizeInBytes, //
					&beta, //
					m_dstTensorDesc[0], //
					deviceOutData //
					));
	}
	hipdnnSetStream(m_cudnnHandle[1],streams[1]);

	for(int i=0;i<4;i++){
		checkCUDNN(hipdnnConvolutionForward(m_cudnnHandle[1], //
					&alpha, //
					m_srcTensorDesc[1], //
					deviceInData1, //the input data of current layer ,also is the output data of preLayer.
					m_filterDesc[1], //
					deviceWeight1, //
					m_convDesc[1], //
					algo, //
					m_fwdDataWorkSpace1, //
					m_fwdDataSizeInBytes, //
					&beta, //
					m_dstTensorDesc[1], //
					deviceOutData1 //
					));
	}

	hipDeviceSynchronize();

	checkCudaErrors(hipFree(deviceInData));
	checkCudaErrors(hipFree(deviceOutData));
	checkCudaErrors(hipFree(deviceWeight));
	checkCudaErrors(hipFree(m_fwdDataWorkSpace));


	checkCudaErrors(hipFree(deviceInData1));
	checkCudaErrors(hipFree(deviceOutData1));
	checkCudaErrors(hipFree(deviceWeight1));
	checkCudaErrors(hipFree(m_fwdDataWorkSpace1));

	checkCUDNN(hipdnnDestroy(m_cudnnHandle[0]));
	checkCUDNN(hipdnnDestroyConvolutionDescriptor(m_convDesc[0]));
	checkCUDNN(hipdnnDestroyFilterDescriptor(m_filterDesc[0]));
	checkCUDNN(hipdnnDestroyTensorDescriptor(m_srcTensorDesc[0]));
	checkCUDNN(hipdnnDestroyTensorDescriptor(m_dstTensorDesc[0]));
	checkCUDNN(hipdnnDestroyTensorDescriptor(m_biasTensorDesc[0]));
	checkCUDNN(hipdnnDestroy(m_cudnnHandle[1]));
	checkCUDNN(hipdnnDestroyConvolutionDescriptor(m_convDesc[1]));
	checkCUDNN(hipdnnDestroyFilterDescriptor(m_filterDesc[1]));
	checkCUDNN(hipdnnDestroyTensorDescriptor(m_srcTensorDesc[1]));
	checkCUDNN(hipdnnDestroyTensorDescriptor(m_dstTensorDesc[1]));
	checkCUDNN(hipdnnDestroyTensorDescriptor(m_biasTensorDesc[1]));

	hipStreamDestroy(streams[0]);
	hipStreamDestroy(streams[1]);

}

int m_convAlgorithm;
hipdnnDataType_t m_dataType;
hipdnnTensorFormat_t m_tensorFormat;
hipdnnHandle_t m_cudnnHandle[2];
hipdnnTensorDescriptor_t m_srcTensorDesc[2], m_dstTensorDesc[2], m_biasTensorDesc[2];
hipdnnFilterDescriptor_t m_filterDesc[2];
hipdnnConvolutionDescriptor_t m_convDesc[2];
double* deviceInData;
double* deviceOutData;
double* deviceWeight;

double* deviceInData1;
double* deviceOutData1;
double* deviceWeight1;
void * m_fwdDataWorkSpace;
void * m_fwdDataWorkSpace1;

void testConnvOnDiffDevice(){



	m_dataType = HIPDNN_DATA_DOUBLE;
	m_convAlgorithm = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;
	m_tensorFormat = HIPDNN_TENSOR_NCHW;

//	pthread_t t1,t2;
//	int ret;
//	ret = pthread_create(&t1, NULL, func3, NULL);
//	if (ret != 0) {
//		LOG(INFO)<<"Error";
//	}
//
//	ret = pthread_create(&t2, NULL, func4, NULL);
//	if (ret != 0) {
//		LOG(INFO)<<"Error";
//	}
//
//	pthread_join(t1,NULL);
//	pthread_join(t2,NULL);

	hipSetDevice(0);
	checkCUDNN(hipdnnCreate(&m_cudnnHandle[0]));
	checkCUDNN(hipdnnCreateTensorDescriptor(&m_srcTensorDesc[0]));
	checkCUDNN(hipdnnCreateTensorDescriptor(&m_dstTensorDesc[0]));
	checkCUDNN(hipdnnCreateTensorDescriptor(&m_biasTensorDesc[0]));
	checkCUDNN(hipdnnCreateFilterDescriptor(&m_filterDesc[0]));
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&m_convDesc[0]));

	hipSetDevice(1);
	checkCUDNN(hipdnnCreate(&m_cudnnHandle[1]));
	checkCUDNN(hipdnnCreateTensorDescriptor(&m_srcTensorDesc[1]));
	checkCUDNN(hipdnnCreateTensorDescriptor(&m_dstTensorDesc[1]));
	checkCUDNN(hipdnnCreateTensorDescriptor(&m_biasTensorDesc[1]));
	checkCUDNN(hipdnnCreateFilterDescriptor(&m_filterDesc[1]));
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&m_convDesc[1]));

	int batchSize = 100;
	int kWidth = 3, kHeight = 3;
	int inMaps = 256, outMaps = 384;
	int inMapSize = 13, outMapSize = 13;

	int n, c, h, w;


	n = batchSize;
	c = inMaps;
	h = inMapSize;
	w = inMapSize;
	hipSetDevice(0);
	SetTensorDesc(m_srcTensorDesc[0], m_dataType, n, c, h, w);
	hipMalloc(&deviceInData, n * c * h * w * sizeof(double));

	hipSetDevice(1);
	SetTensorDesc(m_srcTensorDesc[1], m_dataType, n, c, h, w);
	hipMalloc(&deviceInData1, n * c * h * w * sizeof(double));

	c = outMaps;
	h = outMapSize;
	w = outMapSize;

	const int tensorDims = 4;
	const int filterDimA[tensorDims] = { outMaps, inMaps, kWidth, kHeight };
	const int convDims = 2;
	int padA[convDims] = { 0, 0 };
	int filterStrideA[convDims] = { 1, 1 };
	int upscaleA[convDims] = { 1, 1 };
	long weightSize = kWidth * kHeight * outMaps * inMaps;

	size_t m_fwdDataSizeInBytes;

	hipdnnConvolutionFwdAlgo_t algo = (hipdnnConvolutionFwdAlgo_t) m_convAlgorithm;

	hipSetDevice(0);
	SetTensorDesc(m_dstTensorDesc[0], m_dataType, n, c, h, w);
	hipMalloc(&deviceOutData, n * c * h * w * sizeof(double));
	checkCUDNN(hipdnnSetFilterNdDescriptor(m_filterDesc[0], m_dataType, HIPDNN_TENSOR_NCHW, tensorDims, filterDimA));
	checkCUDNN(hipdnnSetConvolutionNdDescriptor(m_convDesc[0], convDims, padA, filterStrideA, upscaleA, HIPDNN_CROSS_CORRELATION, m_dataType));
	hipMalloc(&deviceWeight, weightSize * sizeof(double));

	hipSetDevice(1);
	SetTensorDesc(m_dstTensorDesc[1], m_dataType, n, c, h, w);
	hipMalloc(&deviceOutData1, n * c * h * w * sizeof(double));
	checkCUDNN(hipdnnSetFilterNdDescriptor(m_filterDesc[1], m_dataType, HIPDNN_TENSOR_NCHW, tensorDims, filterDimA));
	checkCUDNN(hipdnnSetConvolutionNdDescriptor(m_convDesc[1], convDims, padA, filterStrideA, upscaleA, HIPDNN_CROSS_CORRELATION, m_dataType));
	hipMalloc(&deviceWeight1, weightSize * sizeof(double));

	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(m_cudnnHandle[0], m_srcTensorDesc[0], m_filterDesc[0], m_convDesc[0], m_dstTensorDesc[0], algo, &m_fwdDataSizeInBytes));
	if (m_fwdDataSizeInBytes != 0) {
		hipSetDevice(0);
		checkCudaErrors(hipMalloc(&m_fwdDataWorkSpace, m_fwdDataSizeInBytes));
		hipSetDevice(1);
		checkCudaErrors(hipMalloc(&m_fwdDataWorkSpace1, m_fwdDataSizeInBytes));
	}
//	hipStream_t streams[2];
//	for (int i = 0; i < 2; i++)
//	{
//		checkCudaErrors(hipStreamCreate(&(streams[i])));
//	}


	double alpha = 1.0;
	double beta = 0.0;
	//hipdnnSetStream(m_cudnnHandle[0],streams[0]);
	hipSetDevice(0);

	for(int i=0;i<4;i++){
		checkCUDNN(hipdnnConvolutionForward(m_cudnnHandle[0], //
					&alpha, //
					m_srcTensorDesc[0], //
					deviceInData, //the input data of current layer ,also is the output data of preLayer.
					m_filterDesc[0], //
					deviceWeight, //
					m_convDesc[0], //
					algo, //
					m_fwdDataWorkSpace, //
					m_fwdDataSizeInBytes, //
					&beta, //
					m_dstTensorDesc[0], //
					deviceOutData //
					));
	}
	//hipdnnSetStream(m_cudnnHandle[1],streams[1]);
	hipSetDevice(1);
	for(int i=0;i<4;i++){
		checkCUDNN(hipdnnConvolutionForward(m_cudnnHandle[1], //
					&alpha, //
					m_srcTensorDesc[1], //
					deviceInData1, //the input data of current layer ,also is the output data of preLayer.
					m_filterDesc[1], //
					deviceWeight1, //
					m_convDesc[1], //
					algo, //
					m_fwdDataWorkSpace1, //
					m_fwdDataSizeInBytes, //
					&beta, //
					m_dstTensorDesc[1], //
					deviceOutData1 //
					));
	}
	hipDeviceSynchronize();

//	pthread_t t3,t4;
//
//	ret = pthread_create(&t3, NULL, func5, NULL);
//	if (ret != 0) {
//		LOG(INFO)<<"Error";
//	}
//
//	ret = pthread_create(&t4, NULL, func6, NULL);
//	if (ret != 0) {
//		LOG(INFO)<<"Error";
//	}
//
//	pthread_join(t1,NULL);
//	pthread_join(t2,NULL);

	hipSetDevice(0);
	checkCudaErrors(hipFree(deviceInData));
	checkCudaErrors(hipFree(deviceOutData));
	checkCudaErrors(hipFree(deviceWeight));
	checkCudaErrors(hipFree(m_fwdDataWorkSpace));

	checkCUDNN(hipdnnDestroy(m_cudnnHandle[0]));
	checkCUDNN(hipdnnDestroyConvolutionDescriptor(m_convDesc[0]));
	checkCUDNN(hipdnnDestroyFilterDescriptor(m_filterDesc[0]));
	checkCUDNN(hipdnnDestroyTensorDescriptor(m_srcTensorDesc[0]));
	checkCUDNN(hipdnnDestroyTensorDescriptor(m_dstTensorDesc[0]));
	checkCUDNN(hipdnnDestroyTensorDescriptor(m_biasTensorDesc[0]));

	hipSetDevice(1);
	checkCudaErrors(hipFree(deviceInData1));
	checkCudaErrors(hipFree(deviceOutData1));
	checkCudaErrors(hipFree(deviceWeight1));
	checkCudaErrors(hipFree(m_fwdDataWorkSpace1));

	checkCUDNN(hipdnnDestroy(m_cudnnHandle[1]));
	checkCUDNN(hipdnnDestroyConvolutionDescriptor(m_convDesc[1]));
	checkCUDNN(hipdnnDestroyFilterDescriptor(m_filterDesc[1]));
	checkCUDNN(hipdnnDestroyTensorDescriptor(m_srcTensorDesc[1]));
	checkCUDNN(hipdnnDestroyTensorDescriptor(m_dstTensorDesc[1]));
	checkCUDNN(hipdnnDestroyTensorDescriptor(m_biasTensorDesc[1]));

//	hipStreamDestroy(streams[0]);
//	hipStreamDestroy(streams[1]);

}

void testD2DOnDiffDevice(){

	double startTime,endTime;
	long size=1024*1024*10;
	long sizeC=1024*1024*500;
	double* deviceA=NULL;
	double* deviceB=NULL;
	double* deviceA2=NULL;
	double* deviceB2=NULL;
	int* deviceC=NULL;

	int nStreams=3;
	hipStream_t streams[nStreams];
	hipblasHandle_t cublasHandle;

	double *d_A,*d_B,*d_C;
    //fc7
    int M = 512;  //batch_size
	int N = 4096; //in
	int Q = 4096; //out

	long size_A = (M * N); //input_data
	long size_B = (N * Q); //weight
	long size_C = (M * Q); //output_data

	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipMalloc((void**)&deviceA,size*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&deviceA2,size*sizeof(double)));

	checkCudaErrors(hipSetDevice(1));
	for(int i=0;i<nStreams;++i){
		hipStreamCreate(&streams[i]);
	}
	checkCublasErrors(hipblasCreate(&cublasHandle));
	checkCudaErrors(hipMalloc((void**)&deviceB,size*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&deviceB2,size*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&deviceC,sizeC*sizeof(int)));

	checkCudaErrors(hipMalloc(&d_A, size_A * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_B, size_B * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_C, size_C * sizeof(double)));

	startTime = omp_get_wtime();
	//checkCudaErrors(hipMemcpyAsync(deviceB,deviceA,size*sizeof(double),hipMemcpyDeviceToDevice,streams[0]));
	checkCudaErrors(hipMemcpyAsync(deviceB2,deviceA2,size*sizeof(double),hipMemcpyDeviceToDevice,streams[1]));
	double alpha = 1.0;
	double beta = 0.0;
    checkCublasErrors(hipblasSetStream(cublasHandle,streams[2]));
    checkCublasErrors(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, Q, M, N, &alpha, d_B, Q, d_A, N, &beta, d_C, Q));
	//myKernel<<<1,1,0,streams[2]>>>(deviceC);
    hipDeviceSynchronize();
	endTime = omp_get_wtime();
	cout << "Runtime3:" << endTime - startTime << endl;

	hipSetDevice(0);
	hipFree(deviceA);
	hipFree(deviceA2);

	hipSetDevice(1);
	for(int i=0;i<nStreams;++i){
		hipStreamDestroy(streams[i]);
	}
	 checkCublasErrors(hipblasDestroy(cublasHandle));
	hipFree(deviceB2);
	hipFree(deviceB);
	hipFree(deviceC);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipDeviceReset();
}

int main(int argc, char *argv[]) {

	//testKernelWithStreams();
	//testGemmsOnDiffDevice();
	//testGemmsOnSameDevice();
	//testConnvOnSameDevice();
	//testConnvOnDiffDevice();
	//testD2DOnDiffDevice();
	//testDgemm();

	hipStream_t* stream=NULL;
	assert(stream==NULL);
	return 0;
}

